#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "err.h"
#include "ops.h"

namespace ops {

float sum(const float *in, const int size, hipdnnHandle_t cudnn_handle) {
  hipdnnTensorDescriptor_t in_desc, out_desc;
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&in_desc));
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&out_desc));
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, size));
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1));

  hipdnnReduceTensorDescriptor_t reduce_desc;
  CUDNN_CHECK(hipdnnCreateReduceTensorDescriptor(&reduce_desc));
  CUDNN_CHECK(hipdnnSetReduceTensorDescriptor(
        reduce_desc, 
        HIPDNN_REDUCE_TENSOR_ADD, 
        HIPDNN_DATA_FLOAT, 
        HIPDNN_NOT_PROPAGATE_NAN, 
        HIPDNN_REDUCE_TENSOR_NO_INDICES, 
        HIPDNN_32BIT_INDICES));

  size_t workspace_size;
  CUDNN_CHECK(hipdnnGetReductionWorkspaceSize(cudnn_handle, reduce_desc, in_desc, out_desc, &workspace_size));
  size_t indices_size;
  CUDNN_CHECK(cudnnGetReductionIndicesSize(cudnn_handle, reduce_desc, in_desc, out_desc, &indices_size));
  void *workspace_buffer;
  CUDA_CHECK(hipMalloc(&workspace_buffer, workspace_size));
  void *indices_buffer;
  CUDA_CHECK(hipMalloc(&indices_buffer, indices_size));

  float alpha = 1.0f;
  float beta = 0.0f;

  // reduce
  float *result;
  hipMalloc(&result, sizeof(float));
  CUDNN_CHECK(hipdnnReduceTensor(
        cudnn_handle,
        reduce_desc,
        indices_buffer,
        indices_size,
        workspace_buffer,
        workspace_size,
        &alpha,
        in_desc,
        in,
        &beta,
        out_desc,
        result));

  float ret;
  CUDA_CHECK(hipMemcpy(&ret, result, sizeof(float), hipMemcpyDeviceToHost));

  // clean-up
  CUDNN_CHECK(hipdnnDestroyTensorDescriptor(in_desc));
  CUDNN_CHECK(hipdnnDestroyTensorDescriptor(out_desc));
  CUDNN_CHECK(hipdnnDestroyReduceTensorDescriptor(reduce_desc));
  CUDA_CHECK(hipFree(workspace_buffer));
  CUDA_CHECK(hipFree(indices_buffer));
  CUDA_CHECK(hipFree(result));

  return ret;
}

} // namespace ops
